#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "rwN04Fr1Periodic.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

rwN04Fr1Periodic::rwN04Fr1Periodic(GlobalData *_gdata) : Problem(_gdata)
{
	// *** user parameters from command line
	// const DensityDiffusionType RHODIFF = get_option("density-diffusion", DELTA_SPH);
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", BREZZI);
	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	// const int RHODIFF = get_option("density-diffusion", 3);
	const uint ppH = get_option("ppH", 25);
	const bool USE_CCSPH = get_option("use_ccsph", true);

	// use planes in general
	const bool use_planes = get_option("use_planes", false);
	// use a plane for the bottom
	const bool use_bottom_plane = get_option("bottom-plane", use_planes);
	if (use_bottom_plane && !use_planes)
		throw std::invalid_argument("cannot use bottom plane if not using planes");

	// *** Framework setup
	SETUP_FRAMEWORK(
		space_dimensions<R2>,
		periodicity<PERIODIC_X>,
		turbulence_model<LAMINAR_FLOW>,
		// computational_visc<DYNAMIC>,
		// viscosity<DYNAMICVISC>,
		rheology<POWER_LAW>,
		// boundary<DUMMY_BOUNDARY>
		boundary<LJ_BOUNDARY>,
		add_flags<ENABLE_REPACKING>
		// add_flags<ENABLE_INTERNAL_ENERGY>
	).select_options(
		RHODIFF,
		use_planes, add_flags<ENABLE_PLANES>(),
		USE_CCSPH, add_flags<ENABLE_CCSPH>()
	);

	const int mlsIters = get_option("mls",
		(simparams()->densitydiffusiontype != DENSITY_DIFFUSION_NONE) ? 0 : 10);
	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	addPostProcess(SURFACE_DETECTION);

	nd = 0.00209955189; // normal depth
	nv = 0.143385832; // normal velocity
	wl = nd*48.911; // wavelength

	distAmp = 0.225;

	lx = wl;
	ly = 3.0*nd;

	grav = 9.81;

	channelSin = 0.06;
	mudRho = 1120.0;
	powerLawN = 0.40;
	mudMu = 0.14;

	set_deltap(nd/ppH);

	// m_size = make_double3(lx, ly, 1.0);

	setMaxFall(ly);

	// SPH parameters
	simparams()->dtadaptfactor = 0.1;
	simparams()->tend = 32.0;
	simparams()->buildneibsfreq = 10;
	// simparams()->ferrariLengthScale = H;
	simparams()->densityDiffCoeff = 0.05f;

	// Repacking options
	simparams()->repack_maxiter = 10;
	simparams()->repack_a = 0.1;
	simparams()->repack_alpha = 0.1;

	// dyn_thickness = 3.0*m_deltap;

	// Physical parameters
	set_gravity(make_float3(channelSin*grav, (-1.0)*(pow((1-channelSin*channelSin),0.50))*grav, 0.0));
	// g = get_gravity_magnitude();
	// purely for cosmetic reason, let's round the soundspeed to the next
	// integer
	const float c0 = 100.0*nv;
	// auto mud = add_fluid(mudRho);
	add_fluid(mudRho);
	// Surge speed
	setMaxParticleSpeed(5.0*nv);
	// set_equation_of_state(0, 7.0f, c0);
	set_equation_of_state(0, 7.0f, NAN);

	set_consistency_index(0, (mudMu*pow(pow(2.0,0.5),(powerLawN-1.0))));
	set_visc_power_law(0, powerLawN);
	// physparams()->artvisccoeff = 1e-6*10.0/(physparams()->sscoeff[0]*simparams()->slength);

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);
	add_writer(COMMONWRITER, 0.01);

	// Name of problem used for directory creation
	m_name = "rwN04Fr1PeriodicMini";

	setFillingMethod(Object::BORDER_TANGENT);

	// Building the geometry
	setPositioning(PP_CORNER);
	
	// const int num_layers = (simparams()->boundarytype > SA_BOUNDARY) ?
	// 	simparams()->get_influence_layers() : 1;
	// const double box_thickness = (num_layers - 1)*m_deltap;
	// setDynamicBoundariesLayers(num_layers);
	// // place the walls: as planes, if required; otherwise, as boxes
	// if (use_planes) {
	// 	addPlane(0.0, 1.0, 0.0, 0.0); //bottom plane
	// } else {
	// 	// flat bottom rectangle (before the slope begins)
	// 	GeometryID bottom = addRect(GT_FIXED_BOUNDARY, FT_SOLID,
	// 		Point(paddle_origin - make_double3(box_thickness, box_thickness, 0)),
	// 		h_length + box_thickness + rot_correction, box_thickness);
	// 	setUnfillRadius(bottom, 0.5*m_deltap);
	// }

	const double half_dp = 0.5*m_deltap;
	GeometryID domain_box = addRect(GT_FIXED_BOUNDARY, FT_SOLID,
			Point(half_dp, -3.5*m_deltap, 0), lx-m_deltap, 3*m_deltap);

	// const double half_dp = 0.5*m_deltap;
	// GeometryID domain_box = addRect(GT_FIXED_BOUNDARY, FT_SOLID,
			// Point(half_dp, -3.5*m_deltap, 0), l-m_deltap, 3*m_deltap);

	double3 m_fluidOrigin = make_double3(half_dp, half_dp, 0.0);

	GeometryID fluid = addRect(GT_FLUID, FT_SOLID,
		m_fluidOrigin, lx - m_deltap, nd);
	// double3 m_fluidOrigin = make_double3(0.0, 0.0, 0.0);

	// for now. don't know how to create a complicated free-surface IC
	// GeometryID fluidBox = addRect(GT_FLUID, FT_SOLID,
	// 	m_fluidOrigin, lx, nd);
}

void rwN04Fr1Periodic::initializeParticles(BufferList &buffer, const uint numParticle)
	{


		double4 *gpos = buffer.getData<BUFFER_POS_GLOBAL>();
		float4 *pos = buffer.getData<BUFFER_POS>();
		float4 *vel = buffer.getData<BUFFER_VEL>();
		const ushort4 *pinfo = buffer.getData<BUFFER_INFO>();

		for (uint i = 0 ; i < numParticle ; i++) {
			if (FLUID(pinfo[i])){

				double4 pg = gpos[i];
				// vel[i].x = nv*(1.0+distAmp*sin(2.0*M_PI*pg.x/wl));
				// vel[i].x = (1.0+2.0*powerLawN)/(1.0+powerLawN)*(nv*(1.0+distAmp*sin(2.0*M_PI*pg.x/wl)))*(1.0-(pow((1.0-pg.y/nd),((1.0+powerLawN)/powerLawN))));
				vel[i].x = 0.0f;
				vel[i].y = 0.0;
				pos[i].w = physical_density(vel[i].w, 0)*m_deltap*m_deltap;
			}
		}
	}

void rwN04Fr1Periodic::fillDeviceMap()
	{
		fillDeviceMapByAxis(X_AXIS);
	}


